#include "hip/hip_runtime.h"
#include "video_input.h"

#define THREAD_COUNT 16
#define DIVCEIL(X, Y) ((X) + (Y) - 1) / (Y)

Window* ssbml::video_input::get_all_visible_windows(Display **display,
  unsigned long *numWindows)
{
  *display = XOpenDisplay(NULL);
  Atom property = XInternAtom(*display, "_NET_CLIENT_LIST", False), type;
  int form;
  unsigned long remaining;
  unsigned char *list;

  errno = 0;
  if (XGetWindowProperty(*display, XDefaultRootWindow(*display), property, 0,
    1024, False, XA_WINDOW, &type, &form, numWindows, &remaining, &list)
    != Success)
  {
    perror("ssbml::video_input::get_all_visible_windows(): " \
      "XGetWindowProperty()");
    return NULL;
  }

  return (Window*)list;
}

uint32_t ssbml::video_input::get_window_pid(Display *display, Window window)
{
  Atom property = XInternAtom(display, "_NET_WM_PID", False), type;
  uint32_t *pidContainer, pid;
  int form;
  unsigned long remaining, bytes;
  if (XGetWindowProperty(display, window, property, 0, 1, False, XA_CARDINAL,
    &type, &form, &bytes, &remaining, (unsigned char**)(&pidContainer))
    != Success)
  {
    perror("ssbml::video_input::get_window_pid(): XGetWindowProperty()");
    return 0;
  }
  pid = *pidContainer;
  XFree(pidContainer);
  return pid;
}

std::string ssbml::video_input::get_process_name(uint32_t pid)
{
  std::string processName;
  std::ifstream cmdline("/proc/" + std::to_string(pid) + "/cmdline");
  std::getline(cmdline, processName);
  cmdline.close();
  return processName;
}

std::string ssbml::video_input::get_window_title(Display *display,
  Window window)
{
  char *windowTitle;
  if (XFetchName(display, window, &windowTitle))
  {
    std::string tmp(windowTitle);
    XFree(windowTitle);
    return std::string(tmp);
  }
  return "";
}

ssbml::video_input::video_input(Display *display, Window window,
  uint64_t frameWidth, uint64_t frameHeight) :
  display(display),
  window(window),
  frameWidth(frameWidth),
  frameHeight(frameHeight),
  frameSize(frameWidth * frameHeight),
  chromaSize(frameWidth * frameHeight / 4),
  xcbCon(xcb_connect(DisplayString(display), NULL))
{
  if (xcb_connection_has_error(xcbCon) > 0)
  {
    throw std::runtime_error("Could not connect to X server");
  }

  XGetWindowAttributes(display, window, &windowAttributes);

  hipMalloc(&rgbaBuf1, windowAttributes.width * windowAttributes.height * 4);
  hipMalloc(&rgbaBuf2, windowAttributes.width * windowAttributes.height * 4);
  hipMalloc(&rgbBuf, windowAttributes.width * windowAttributes.height * 3);
  hipMalloc(&lumaBuf, frameSize);
  hipMalloc(&redChromaBuf, frameSize);
  hipMalloc(&blueChromaBuf, frameSize);
}

ssbml::video_input::~video_input()
{
  xcb_disconnect(xcbCon);
  hipFree(rgbaBuf1);
  hipFree(rgbaBuf2);
  hipFree(rgbBuf);
  hipFree(lumaBuf);
  hipFree(redChromaBuf);
  hipFree(blueChromaBuf);
}

__global__ void reduce_width(uint8_t *rgbaBuf_in, uint8_t *rgbaBuf_out,
  int width, int height)
{
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int half = width / 2;

  if (col < half && row < height)
  {
    uint32_t p1 = 0, p2 = 0;
    int col2 = col * 2;
    p1 = ((uint32_t*)rgbaBuf_in)[row * width + col2];
    p2 = ((uint32_t*)rgbaBuf_in)[row * width + (col2 + 1)];
    ((uint32_t*)rgbaBuf_out)[row * half + col] = __vavgu4(p1, p2);
  }
}

__global__ void reduce_height(uint8_t *rgbaBuf_in, uint8_t *rgbaBuf_out,
  uint64_t width, uint64_t height)
{
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int half = height / 2;

  if (col < width && row < half)
  {
    uint32_t p1 = 0, p2 = 0;
    int row2 = row * 2;
    p1 = ((uint32_t*)rgbaBuf_in)[row2 * width + col];
    p2 = ((uint32_t*)rgbaBuf_in)[(row2 + 1) * width + col];
    ((uint32_t*)rgbaBuf_out)[row * width + col] = __vavgu4(p1, p2);
  }
}

__device__ float lerp(float s, float e, float t)
{
  return s + (e - s) * t;
}

__device__ float blerp(float c00, float c10, float c01, float c11, float tx, float ty)
{
  return lerp(lerp(c00, c10, tx), lerp(c01, c11, tx), ty);
}

__global__ void rgba2yuv(const uint8_t *rgbaBuf, uint8_t *lumaBuf,
  uint8_t *redChromaBuf, uint8_t *blueChromaBuf, uint64_t width,
  uint64_t height)
{
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  if (col < width && row < height)
  {
    int index = row * width + col;
    int index2 = index * 4;
    lumaBuf[index] = 0.299 * rgbaBuf[index2] + 0.587 * rgbaBuf[index2 + 1]
      + 0.114 * rgbaBuf[index2 + 2];

    int col2 = col * 2, row2 = row * 2;
    if (col2 < width && row2 < height)
    {
      int i00 = row2 * width + col2;
      int i01 = (row2 + 1) * width + col2;
      int i10 = row2 * width + (col2 + 1);
      int i11 = (row2 + 1) * width + (col2 + 1);

      uint32_t pixel = __vavgu4(
        ((uint32_t*)rgbaBuf)[i00],
        __vavgu4(
          ((uint32_t*)rgbaBuf)[i01],
          __vavgu4(
            ((uint32_t*)rgbaBuf)[i10],
            ((uint32_t*)rgbaBuf)[i11]
          )
        )
      );

      uint8_t blue = (uint8_t)(pixel >> 16);
      uint8_t green = (uint8_t)(pixel >> 8);
      uint8_t red = (uint8_t)pixel;

      index = row * (width / 2) + col;
      blueChromaBuf[index] = (-0.168736 * red + -0.331264 * green + 0.5 * blue)
        + 128;
      redChromaBuf[index] = (0.5 * red + -0.418688 * green + -0.081312 * blue)
        + 128;
    }
  }
}

__global__ void rgba2rgb(const uint8_t *rgbaBuf, uint8_t *rgbBuf,
  uint64_t width, uint64_t height)
{
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  if (col < width && row < height)
  {
    int index = row * width + col;
    int index2 = index * 3;
    uint32_t pixel = ((uint32_t*)rgbaBuf)[index];
    uint8_t red = (uint8_t)(pixel >> 16);
    uint8_t green = (uint8_t)(pixel >> 8);
    uint8_t blue = (uint8_t)pixel;

    rgbBuf[index2] = red;
    rgbBuf[index2 + 1] = green;
    rgbBuf[index2 + 2] = blue;
  }
}

// rgbaBuf initially has rgba but will end with just rgb
__global__ void bilinear(uint8_t *rgbaBuf_in, uint8_t *rgbaBuf_out,
  uint64_t newWidth, uint64_t newHeight, uint64_t oldWidth, uint64_t oldHeight)
{
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int in = col < newWidth && row < newHeight;
  uint8_t red = 0, green = 0, blue = 0;
  if (in)
  {
    float gy = row / (float)newHeight * (oldHeight - 1);
    int gyi = (int)gy;
    float gx = col / (float)newWidth * (oldWidth - 1);
    int gxi = (int)gx;
    float dx = gx - gxi, dy = gy - gyi;

    uint8_t *p00 = rgbaBuf_in + (gyi * oldWidth + gxi) * 4;
    uint8_t *p10 = rgbaBuf_in + (gyi * oldWidth + gxi + 1) * 4;
    uint8_t *p01 = rgbaBuf_in + ((gyi + 1) * oldWidth + gxi) * 4;
    uint8_t *p11 = rgbaBuf_in + ((gyi + 1) * oldWidth + gxi + 1) * 4;

    red = (uint8_t)blerp(p00[2], p10[2], p01[2], p11[2], dx, dy);
    green = (uint8_t)blerp(p00[1], p10[1], p01[1], p11[1], dx, dy);
    blue = (uint8_t)blerp(p00[0], p10[0], p01[0], p11[0], dx, dy);

    int index = (row * newWidth + col) * 4;
    rgbaBuf_out[index] = red;
    rgbaBuf_out[index + 1] = green;
    rgbaBuf_out[index + 2] = blue;
    rgbaBuf_out[index + 3] = 255;
  }
}

void ssbml::video_input::get_image()
{
  xcb_get_image_cookie_t cookie;
  xcb_get_image_reply_t *reply;
  xcb_generic_error_t *err = NULL;
  uint64_t width = windowAttributes.width, height = windowAttributes.height;
  uint8_t *imageData;

  cookie = xcb_get_image(xcbCon, XCB_IMAGE_FORMAT_Z_PIXMAP, window, 0, 0, width, height, ~0);
  reply = xcb_get_image_reply(xcbCon, cookie, &err);
  if (err)
  {
    std::cerr << "Cannot grab window image data. response type="
      << err->response_type << " error code=" << err->error_code << " sequence="
      << err->sequence << " resource id=" << err->resource_id << " minor code="
      << err->minor_code << " major code=" << err->major_code << std::endl;
  }
  if (!reply)
  {
    std::cerr << "Reply was empty" << std::endl;
  }
  imageData = xcb_get_image_data(reply);
  hipMemcpy(rgbaBuf1, imageData, height * width * 4, hipMemcpyHostToDevice);
  free(reply);

  /*XImage *image = XGetImage(display, window, 0, 0, width, height, AllPlanes,
    ZPixmap);
  hipMemcpy(rgbaBuf1, image->data, height * width * 4, hipMemcpyHostToDevice);
  XFree(image);*/

  uint8_t *rgbaBuf_in = rgbaBuf1, *rgbaBuf_out = rgbaBuf2;
  dim3 threads(THREAD_COUNT, THREAD_COUNT);
  while (width / 2 >= frameWidth)
  {
    reduce_width<<<dim3(DIVCEIL(width / 2, THREAD_COUNT),
      DIVCEIL(height, THREAD_COUNT)), threads>>>(rgbaBuf_in, rgbaBuf_out, width,
      height);
    hipDeviceSynchronize();
    width /= 2;
    std::swap(rgbaBuf_in, rgbaBuf_out);
  }
  while (height / 2 >= frameHeight)
  {
    reduce_height<<<dim3(DIVCEIL(width, THREAD_COUNT),
      DIVCEIL(height / 2, THREAD_COUNT)), threads>>>(rgbaBuf_in, rgbaBuf_out,
      width, height);
    hipDeviceSynchronize();
    height /= 2;
    std::swap(rgbaBuf_in, rgbaBuf_out);
  }

  if (height != frameHeight && width != frameWidth)
  {
    dim3 blocks(DIVCEIL(frameWidth, THREAD_COUNT),
      DIVCEIL(frameHeight, THREAD_COUNT));
    bilinear<<<blocks, threads>>>(rgbaBuf_in, rgbaBuf_out, frameWidth,
      frameHeight, width, height);
    std::swap(rgbaBuf_in, rgbaBuf_out);
    hipDeviceSynchronize();
  }
  rgbaBuf = rgbaBuf_in;
}

ssbml::video_input& ssbml::video_input::operator>>(video_output &videoOutput)
{
  get_frame(*videoOutput.frame);
  videoOutput.write_frame();
  return *this;
}

ssbml::video_input& ssbml::video_input::operator>>(uint8_t *rgbBuf)
{
  get_frame(rgbBuf);
  return *this;
}

void ssbml::video_input::get_frame(uint8_t *rgbBuf)
{
  get_image();
  dim3 threads(THREAD_COUNT, THREAD_COUNT);
  dim3 blocks(DIVCEIL(frameWidth, THREAD_COUNT),
    DIVCEIL(frameHeight, THREAD_COUNT));
  rgba2rgb<<<blocks, threads>>>(rgbaBuf, this->rgbBuf, frameWidth, frameHeight);
  hipDeviceSynchronize();

  hipMemcpy(rgbBuf, this->rgbBuf, frameSize * 3, hipMemcpyDeviceToHost);
}

void ssbml::video_input::get_frame(AVFrame &frame)
{
  get_image();
  dim3 threads(THREAD_COUNT, THREAD_COUNT);
  dim3 blocks((frameWidth + THREAD_COUNT - 1 ) / THREAD_COUNT,
    (frameHeight + THREAD_COUNT - 1) / THREAD_COUNT);
  rgba2yuv<<<blocks, threads>>>(rgbaBuf, lumaBuf, redChromaBuf, blueChromaBuf,
    frameWidth, frameHeight);
  hipDeviceSynchronize();

  av_frame_make_writable(&frame);
  hipMemcpy(frame.data[0], lumaBuf, frameSize, hipMemcpyDeviceToHost);
  hipMemcpy(frame.data[1], blueChromaBuf, chromaSize, hipMemcpyDeviceToHost);
  hipMemcpy(frame.data[2], redChromaBuf, chromaSize, hipMemcpyDeviceToHost);
}
