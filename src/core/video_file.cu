#include "hip/hip_runtime.h"
#include "video_file.h"

#define THREAD_COUNT 16
#define DIVCEIL(X, Y) ((X) + (Y) - 1) / (Y)

ssbml::video_file::~video_file()
{
  av_frame_free(&frame);
  av_packet_free(&pkt);
  avcodec_close(cctx);
  avformat_close_input(&fctx);

  hipFree(rgbBuf);
  hipFree(lumaBuf);
  hipFree(redChromaBuf);
  hipFree(blueChromaBuf);
}

ssbml::video_file::video_file(std::string fileName) : nextFrame(0), fctx(NULL),
  cctx(NULL), codec(NULL), frame(NULL)
{
  if (avformat_open_input(&fctx, fileName.c_str(), NULL, NULL) != 0)
  {
    throw std::runtime_error("Failed to open file");
  }
  if (avformat_find_stream_info(fctx, NULL) < 0)
  {
    avformat_close_input(&fctx);
    throw std::runtime_error("Couldn't find stream information");
  }

  av_dump_format(fctx, 0, fileName.c_str(), 0);

  AVStream *stream = nullptr;
  for (uint64_t i = 0; i < fctx->nb_streams; ++i)
  {
    if (fctx->streams[i]->codecpar->codec_type == AVMEDIA_TYPE_VIDEO)
    {
      stream = fctx->streams[i];
      break;
    }
  }
  if (stream == nullptr)
  {
    avformat_close_input(&fctx);
    avcodec_close(cctx);
    throw std::runtime_error("File does not contain a video stream");
  }
  if ((codec = avcodec_find_decoder(stream->codecpar->codec_id)) == NULL)
  {
    avformat_close_input(&fctx);
    throw std::runtime_error("File uses unsupported codec");
  }
  if ((cctx = avcodec_alloc_context3(codec)) == NULL)
  {
    avformat_close_input(&fctx);
    throw std::runtime_error("Failed to allocate AVCodecContext");
  }
  if (avcodec_parameters_to_context(cctx, stream->codecpar) < 0)
  {
    avformat_close_input(&fctx);
    throw std::runtime_error("Failed to convert codec parameters to a " \
      "codec context");
  }
  uint64_t fps = stream->avg_frame_rate.num / stream->avg_frame_rate.den;
  totalFrames = (fctx->duration * fps) / 1000000;

  if (avcodec_open2(cctx, codec, NULL) < 0)
  {
    avcodec_close(cctx);
    avformat_close_input(&fctx);
    throw std::runtime_error("Failed to open codec");
  }
  if ((frame = av_frame_alloc()) == NULL)
  {
    avcodec_close(cctx);
    avformat_close_input(&fctx);
    throw std::runtime_error("Failed to allocated AVFrame");
  }

  if ((pkt = av_packet_alloc()) == NULL)
  {
    av_frame_free(&frame);
    avcodec_close(cctx);
    avformat_close_input(&fctx);
    throw std::runtime_error("Could not allocate AVPacket");
  }

  frameSize = cctx->width * cctx->height;
  hipMalloc(&rgbBuf, frameSize * 3);
  hipMalloc(&lumaBuf, frameSize);
  hipMalloc(&redChromaBuf, frameSize);
  hipMalloc(&blueChromaBuf, frameSize);
}

uint64_t ssbml::video_file::get_total_frames()
{
  return totalFrames;
}

uint64_t ssbml::video_file::get_frame_width()
{
  return cctx->width;
}

uint64_t ssbml::video_file::get_frame_height()
{
  return cctx->height;
}

ssbml::video_file& ssbml::video_file::operator>>(uint8_t *rgbBuf)
{
  get_frame(rgbBuf);
  return *this;
}

ssbml::video_file& ssbml::video_file::operator>>(video_output &videoOutput)
{
  get_frame(*videoOutput.frame);
  return *this;
}

__global__ void yuv2rgb(uint8_t *rgbBuf, const uint8_t *lumaBuf,
  const uint8_t *redChromaBuf, const uint8_t *blueChromaBuf, uint64_t width,
  uint64_t height)
{
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  if (col < width && row < height)
  {
    int index = row * width + col;
    int index2 = index * 3;
    int chromaIndex = (row / 2) * (width / 2) + (col / 2);

    uint8_t luma = lumaBuf[index];
    uint8_t redChroma = redChromaBuf[chromaIndex];
    uint8_t blueChroma = blueChromaBuf[chromaIndex];

    uint16_t tmp = luma + 1.4075 * (redChroma - 128);
    uint8_t red = tmp > 255 ? 255 : tmp;
    tmp = luma - 0.3455 * (blueChroma - 128) - (0.7169 * (redChroma - 128));
    uint8_t green = tmp > 255 ? 255 : tmp;
    tmp = luma + 1.7790 * (blueChroma - 128);
    uint8_t blue = tmp > 255 ? 255 : tmp;

    rgbBuf[index2] = red;
    rgbBuf[index2 + 1] = green;
    rgbBuf[index2 + 2] = blue;
  }
}

void ssbml::video_file::get_image()
{
  int ret;
  if (nextFrame < totalFrames)
  {
    ret = av_read_frame(fctx, pkt);
    if (ret < 0)
    {
      char errBuf[1024];
      av_strerror(ret, errBuf, sizeof(errBuf));
      throw std::runtime_error("av_read_frame: " + std::string(errBuf));
    }


    ret = avcodec_send_packet(cctx, pkt);
    if (ret < 0)
    {
      char errBuf[1024];
      av_strerror(ret, errBuf, sizeof(errBuf));
      throw std::runtime_error("avcodec_send_packet: " + std::string(errBuf));
    }

    while (ret >= 0)
    {
      ret = avcodec_receive_frame(cctx, frame);
      if (ret == AVERROR(EAGAIN))
      {
        break;
      }
      if (ret < 0)
      {
        char errBuf[1024];
        av_strerror(ret, errBuf, sizeof(errBuf));
        throw std::runtime_error("avcodec_receive_frame: " + std::string(errBuf));
      }
    }

    av_packet_unref(pkt);
    ++nextFrame;
  }
  else
  {
    throw std::runtime_error("Cannot get more frames, end of file");
  }
}

void ssbml::video_file::get_frame(uint8_t *rgbBuf)
{
  get_image();

  hipMemcpy(lumaBuf, frame->data[0], frameSize, hipMemcpyHostToDevice);
  hipMemcpy(blueChromaBuf, frame->data[1], frameSize / 4, hipMemcpyHostToDevice);
  hipMemcpy(redChromaBuf, frame->data[2], frameSize / 4, hipMemcpyHostToDevice);
  dim3 threads(THREAD_COUNT, THREAD_COUNT);
  dim3 blocks(DIVCEIL(cctx->width, THREAD_COUNT),
    DIVCEIL(cctx->height, THREAD_COUNT));
  yuv2rgb<<<blocks, threads>>>(this->rgbBuf, lumaBuf, redChromaBuf, blueChromaBuf,
    cctx->width, cctx->height);
  hipDeviceSynchronize();

  hipMemcpy(rgbBuf, this->rgbBuf, frameSize * 3, hipMemcpyDeviceToHost);
}

void ssbml::video_file::get_frame(AVFrame &frame)
{
  get_image();

  av_frame_make_writable(&frame);
  memcpy(frame.data[0], this->frame->data[0], frameSize);
  memcpy(frame.data[1], this->frame->data[1], frameSize / 4);
  memcpy(frame.data[2], this->frame->data[2], frameSize / 4);
}
